#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include"math.h"

#define TPB 16

#define N 30 // Number of electrons
#define Nk 20 // Number of k-modes
#define Ne 10 // Number of polarizations per k-mode

__constant__ double pi;
__constant__ double q; // electron charge
__constant__ double m; // electron rest mass
//__constant__ double hbar=0.0; // use this to see "classical" results
__constant__ double hbar;
__constant__ double c; // velocity of light in vacuum
__constant__ double eps0;
__constant__ double v0; // electron velocity before laser region
__constant__ double sigma; // electron beam standard deviation
//__constant__ double beta=v0/c;
//__constant__ double gamma=1.0;//pow(1.0-pow(beta,2.0),-0.5);

__constant__ double wC; // Compton frequency
__constant__ double kC; // kC=wC/c
__constant__ double lamC; // lamC=2pi/kC

__constant__ double wL; // Laser frequency
__constant__ double kL; // kL=wL/c
__constant__ double lamL; // lamL=2pi/kL

__constant__ double E0L; // Laser electric field intensity amplitude
__constant__ double D; // Laser beam waist
__constant__ double zimp; // Screen position (origin set right before laser region)

__constant__ double damping; // Damping rate (harmonic oscillator approximation)
__constant__ double Delta; // thickness of the spherical shell in k-space
__constant__ double kmin;
__constant__ double kmax;
__constant__ double dt; // time step necessary to resolve the electron trajectory

void onHost();
void onDevice(double *k,double *theta,double *phi,double *eta,double *xi,double *init,double *pos);
__global__ void setup_kmodes(hiprandState *state,unsigned long seed);
__global__ void kmodes(double *x,hiprandState *state,int option,int n);
__global__ void paths_rk2(double *k,double *theta,double *phi,double *xi,double *eta,double *init,double *pos);
__global__ void paths_rk4(double *k,double *theta,double *phi,double *xi,double *eta,double *init,double *pos);
__device__ double f(double k,double theta,double phi,double xi,double eta1,double eta2,double t,double x,double y,double z,double vy,double vz);
__device__ double g(double k,double theta,double phi,double xi,double eta1,double eta2,double t,double x,double y,double z,double vx,double vz);
__device__ double h(double k,double theta,double phi,double xi,double eta1,double eta2,double t,double x,double y,double z,double vx,double vy);

int main(){
	onHost();
	return 0;
}

void onHost(){
	FILE *k_vec,*posit;

	double *k_h,*theta_h,*phi_h; // Spherical coordinates for each k-mode (Nk in total)
	double *xi_h; // Polarization angles for each k-mode (Ne in total): NOT random
	double *eta_h; // Random phases for the ZPF k-modes (2N in total)
	double *init_h,*pos_h; // Initial and final positions (h indicates host allocation)

	k_h=(double*)malloc(Nk*sizeof(double));
	theta_h=(double*)malloc(Nk*sizeof(double));
	phi_h=(double*)malloc(Nk*sizeof(double));

	xi_h=(double*)malloc(Ne*sizeof(double));

	eta_h=(double*)malloc(2*N*sizeof(double));

	init_h=(double*)malloc(N*sizeof(double));
	pos_h=(double*)malloc(N*sizeof(double));

	onDevice(k_h,theta_h,phi_h,eta_h,xi_h,init_h,pos_h);

	k_vec=fopen("k-vectors.txt","w");
	for(int i=0;i<Nk;i++){
		fprintf(k_vec,"%f,%f,%f\n",k_h[i],theta_h[i],phi_h[i]);
	}
	fclose(k_vec);

	posit=fopen("positions.txt","w");
	for(int i=0;i<N;i++){
		fprintf(posit,"%f,%f\n",init_h[i],pos_h[i]);
	}
	fclose(posit);

	free(k_h);
	free(theta_h);
	free(phi_h);
	free(xi_h);
	free(eta_h);
	free(init_h);
	free(pos_h);
}

void onDevice(double *k_h,double *theta_h,double *phi_h,double *eta_h,double *xi_h,double *init_h,double *pos_h){
	/*const int block_calc=(Nk+TPB-1)/TPB;
	const int blocks=(Nk<block_calc ? 32:block_calc); // Maximum number of resident blocks per SM: 32*/
	unsigned int blocks=(Nk+TPB-1)/TPB;

	double pi_h=3.1415926535;
	double q_h=1.6e-19;
	double m_h=9.10938356e-31;
	double hbar_h=1.0545718e-34;
//	double hbar=0; // uncomment this line to see classical results
	double c_h=299792458.0;
	double eps0_h=8.85e-12;
	double v0_h=1.1e7;
	double fwhm_h=25e-6;
	double sigma_h=fwhm/(2.0*sqrt(2.0*log(2.0)));

	double wC_h=m_h*pow(c_h,2.0)/hbar_h;
	double kC_h=wC_h/c_h;
	double lamC_h=2*pi_h/kC_h;

	double lamL_h=532e-9;
	double kL_h=2*pi_h/lamL_h;
	double wL_h=kL_h*c_h;

	double E0L_h=2.6e8;
	double D_h=125e-6;
	double zimp_h=24e-2+D_h;

	double damping_h=6.245835e-24;
	double Delta_h=1e7*damping_h*pow(wL_h,2.0);
	double kmin_h=(wL_h-Delta_h/2.0)/c_h;
	double kmax_h=(wL_h+Delta_h/2.0)/c_h;
	double dt_h=pi_h/(10.0*(wL_h+Delta_h/2.0));

	hipMemcpyToSymbol(HIP_SYMBOL(pi),&pi_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(q),&q_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(m),&m_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(hbar),&hbar_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c),&c_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(eps0),&eps0_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(v0),&v0_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sigma),&sigma_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(wC),&wC_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kC),&kC_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(lamC),&lamC_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(lamL),&lamL_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kL),&kL_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(wL),&wL_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(E0L),&E0L_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(D),&D_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(zimp),&zimp_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(damping),&damping_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(Delta),&Delta_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kmin),&kmin_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kmax),&kmax_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dt),&dt_h,sizeof(double));

	double *theta_d,*phi_d,*k_d;
	double *xi_d;
	double *eta_d;
	double *pos_d,*init_d; // Vectors in Device (d indicates device allocation)

	printf("Number of particles: %d\n",N);
	printf("Number of k-modes: %d\n",Nk);
	printf("Number of polarizations: %d\n",Ne);
	printf("Threads per block: %d\n",TPB);
	printf("Number of blocks (k-modes): %d\n",blocks);

	hipMalloc((void**)&k_d,Nk*sizeof(double));
	hipMalloc((void**)&theta_d,Nk*sizeof(double));
	hipMalloc((void**)&phi_d,Nk*sizeof(double));

	hipMalloc((void**)&eta_d,2*N*sizeof(double));

	hipMalloc((void**)&xi_d,Ne*sizeof(double));

	hipMalloc((void**)&init_d,N*sizeof(double));
	hipMalloc((void**)&pos_d,N*sizeof(double));

	/* Randomly generated k-modes inside the spherical shell */

	hiprandState *devStates;
        hipMalloc(&devStates,Nk*sizeof(hiprandState));

	//k
	srand(time(0));
	int seed=rand(); //Setting up the seeds
	setup_kmodes<<<blocks,TPB>>>(devStates,seed);

	kmodes<<<blocks,TPB>>>(k_d,devStates,1,Nk);

	//theta
	kmodes<<<blocks,TPB>>>(theta_d,devStates,2,Nk);

	//phi
	kmodes<<<blocks,TPB>>>(phi_d,devStates,3,Nk);

	hipMemcpy(k_h,k_d,Nk*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(theta_h,theta_d,Nk*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(phi_h,phi_d,Nk*sizeof(double),hipMemcpyDeviceToHost);

	/* Randomly generated phases for the CPC modes */

	hiprandState *devStates_n;
	hipMalloc(&devStates_n,2*N*sizeof(hiprandState));

	blocks=(2*N+TPB-1)/TPB;
	printf("Number of blocks (phases): %d\n",blocks);

	//eta
	srand(time(NULL));
	seed=rand(); //Settin up seeds
	setup_kmodes<<<blocks,TPB>>>(devStates_n,seed);

	kmodes<<<blocks,TPB>>>(eta_d,devStates_n,3,2*N);

	hipMemcpy(eta_h,eta_d,Ne*sizeof(double),hipMemcpyDeviceToHost);

	/* Polarization modes allocation (in device memory) */
	for(int i=0;i<Ne;i++){
		xi_h[i]=i*2*pi_h/Ne;
	}

	/* Initial positions */

	blocks=(N+TPB-1)/TPB;
	printf("Number of blocks (paths): %d\n",blocks);

	kmodes<<<blocks,TPB>>>(init_d,devStates_n,4,N);

	paths_rk2<<<blocks,TPB>>>(k_d,theta_d,phi_d,xi_d,eta_d,init_d,pos_d);
	//paths_rk4<<<blocks,TPB>>>(k_d,theta_d,phi_d,xi_d,eta_d,init_d,pos_d);

	hipMemcpy(pos_h,pos_d,N*sizeof(double),hipMemcpyDeviceToHost);

	hipFree(devStates);
	hipFree(devStates_n);
	hipFree(k_d);
	hipFree(theta_d);
	hipFree(phi_d);
	hipFree(xi_d);
	hipFree(eta_d);
	hipFree(init_d);
	hipFree(pos_d);
}

__global__ void setup_kmodes(hiprandState *state,unsigned long seed){
        int idx=threadIdx.x+blockIdx.x*blockDim.x;
        hiprand_init(seed,idx,0,&state[idx]);
}

__global__ void kmodes(double *vec,hiprandState *globalState,int opt,int n){
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	hiprandState localState=globalState[idx];
	if(idx<n){
		if(opt==1){
			vec[idx]=pow((pow(kmax,3.0)-pow(kmin,3.0))*hiprand_uniform(&localState)+pow(kmin,3.0),1.0/3.0); // Random radii
		}else if(opt==2){
			vec[idx]=acos(1.0-2.0*hiprand_uniform(&localState)); // Random polar angles
		}else if(opt==3){
			vec[idx]=2.0*pi*hiprand_uniform(&localState); // Random azimuthal angles
		}else if(opt==4){
			vec[idx]=sigma*hiprand_normal(&localState); // Random initial positions
		}
		globalState[idx]=localState; // Update current seed state
	}
}

__global__ void paths_rk2(double *k,double *theta,double *phi,double *xi,double *eta,double *init,double *pos){
	unsigned int idx=threadIdx.x+blockIdx.x*blockDim.x;
	double tn=0.0;
	double xn=0.0;
	double yn=init[idx];
	double zn=0.0;
	double vxn=0.0;
	double vyn=0.0;
	double vzn=v0;
	double xnn,ynn,znn,vxnn,vynn,vznn;
	double k1vx,k2vx,k1vy,k2vy,k1vz,k2vz;
}

__global__ void paths_rk4(double *k,double *theta,double *phi,double *xi,double *eta,double *init,double *pos){
	unsigned int idx=threadIdx.x+blockIdx.x*blockDim.x;
	double tn=0.0;
	double xn=0.0;
	double yn=init[idx];
	double zn=0.0;
	double vxn=0.0;
	double vyn=0.0;
	double vzn=v0;
	double xnn,ynn,znn,vxnn,vynn,vznn;
	double k1vx,k2vx,k1vy,k2vy,k1vz,k2vz,k3vx,k3vy,k3vz,k4vx,k4vy,k4vz;
}

__device__ double f(double k,double theta,double phi,double xi,double eta1,double eta2,double t,double x,double y,double z,double vy,double vz){

}

__device__ double g(double k,double theta,double phi,double xi,double eta1,double eta2,double t,double x,double y,double z,double vx,double vz){

}

__device__ double h(double k,double theta,double phi,double xi,double eta1,double eta2,double t,double x,double y,double z,double vx,double vy){

}

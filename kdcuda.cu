#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include"math.h"

# define TPB 128

const int N=10000; // Number of electrons
const int Nk=200; // Number of k-modes
const int Ne=10; // Number of polarizations per k-mode

__constant__ int N_d;
__constant__ int Nk_d;
__constant__ int Ne_d;

__constant__ double pi;
__constant__ double q; // electron charge
__constant__ double m; // electron rest mass
//__constant__ double hbar=0.0; // use this to see "classical" results
__constant__ double hbar;
__constant__ double c; // velocity of light in vacuum
__constant__ double eps0;
__constant__ double v0; // electron velocity before laser region
//__constant__ double beta=v0/c;
//__constant__ double gamma=1.0;//pow(1.0-pow(beta,2.0),-0.5);

__constant__ double wC; // Compton frequency
__constant__ double kC; // kC=wC/c
__constant__ double lamC; // lamC=2pi/kC

__constant__ double wL; // Laser frequency
__constant__ double kL; // kL=wL/c
__constant__ double lamL; // lamL=2pi/kL

__constant__ double E0L; // Laser electric field intensity amplitude
__constant__ double D; // Laser beam waist
__constant__ double zimp; // Screen position (origin set right before laser region)

__constant__ double damping; // Damping rate (harmonic oscillator approximation)
__constant__ double Delta; // thickness of the spherical shell in k-space
__constant__ double kmin;
__constant__ double kmax;
__constant__ double dt; // time step necessary to resolve the electron trajectory

void onHost();
void onDevice(double *k,double *theta,double *phi);
__global__ void setup_kmodes(hiprandState *state,unsigned long seed);
__global__ void kmodes(double *x,hiprandState *state,int option);

int main(){
	onHost();
	return 0;
}

void onHost(){
	/*double *init_h,*pos_h; // Initial and final positions (h indicates host allocation)
	double *eta_h; // Random phases for the ZPF k-modes (2N in total)*/
	double *k_h,*theta_h,*phi_h; // Spherical coordinates for each k-mode (Nk in total)
	//double *xi_h; // Random polarization angles for each k-mode (Ne in total)

	/*init_h=(double*)malloc(N);
	pos_h=(double*)malloc(N);
	eta_h=(double*)malloc(2*N);*/

	k_h=(double*)malloc(Nk);
	theta_h=(double*)malloc(Nk);
	phi_h=(double*)malloc(Nk);

	//xi_h=(double*)malloc(Ne);

	onDevice(k_h,theta_h,phi_h);

	/*free(init_h);
	free(pos_h);
	free(eta_h);
	free(k_h);
	free(theta_h);
	free(phi_h);
	free(xi_h);*/

}

void onDevice(double *k_h,double *theta_h,double *phi_h){
	/*const int block_calc=(Nk+TPB-1)/TPB;
	const int blocks=(Nk<block_calc ? 32:block_calc); // Maximum number of resident blocks per SM: 32*/
	const int blocks=(Nk+TPB-1)/TPB;

	double pi_h=3.1415926535;
	double q_h=1.6e-19;
	double m_h=9.10938356e-31;
	double hbar_h=1.0545718e-34;
//	double hbar=0; // uncomment this line to see classical results
	double c_h=299792458.0;
	double eps0_h=8.85e-12;
	double v0_h=1.1e7;

	double wC_h=m_h*pow(c_h,2.0)/hbar_h;
	double kC_h=wC_h/c_h;
	double lamC_h=2*pi_h/kC_h;

	double lamL_h=532e-9;
	double kL_h=2*pi_h/lamL_h;
	double wL_h=kL_h*c_h;

	double E0L_h=2.6e8;
	double D_h=125e-6;
	double zimp_h=24e-2+D_h;

	double damping_h=6.245835e-24;
	double Delta_h=1e7*damping_h*pow(wL_h,2.0);
	double kmin_h=(wL_h-Delta_h/2.0)/c_h;
	double kmax_h=(wL_h+Delta_h/2.0)/c_h;
	double dt_h=pi_h/(10.0*(wL_h+Delta_h/2.0));

	hipMemcpyToSymbol(HIP_SYMBOL(pi),&pi_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(q),&q_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(m),&m_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(hbar),&hbar_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c),&c_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(eps0),&eps0_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(v0),&v0_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(wC),&wC_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kC),&kC_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(lamC),&lamC_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(lamL),&lamL_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kL),&kL_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(wL),&wL_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(E0L),&E0L_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(D),&D_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(zimp),&zimp_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(damping),&damping_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(Delta),&Delta_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kmin),&kmin_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kmax),&kmax_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dt),&dt_h,sizeof(double));

	//double *pos_d,*init_d; // Vectors in Device (d indicates device allocation)
	//double *eta_d;
	double *theta_d,*phi_d,*k_d;
	//double *xi_d;

	printf("Number of particles: %d\n",N);
	printf("Number of k-modes: %d\n",Nk);
	printf("Number of polarizations: %d\n",Ne);
	printf("Threads per block: %d\n",TPB);
	printf("Number of blocks: %d\n",blocks);

	hipMalloc((void**)&k_d,Nk*sizeof(double));
	hipMalloc((void**)&theta_d,Nk*sizeof(double));
	hipMalloc((void**)&phi_d,Nk*sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(N_d),&N,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(Nk_d),&Nk,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(Ne_d),&Ne,sizeof(int));

	/* Randomly generated k-modes inside the spherical shell */

	hiprandState *devStates;
        hipMalloc(&devStates,Nk*sizeof(hiprandState));

	//k
	srand(time(0));
	int seed=rand(); //Setting up the seeds
	setup_kmodes<<<blocks,TPB>>>(devStates,seed);

	kmodes<<<blocks,TPB>>>(k_d,devStates,1);

	//theta
        srand(time(0));
        seed=rand();
        setup_kmodes<<<blocks,TPB>>>(devStates,seed);

	kmodes<<<blocks,TPB>>>(theta_d,devStates,2);

	//phi
	srand(time(0));
	seed=rand();
	setup_kmodes<<<blocks,TPB>>>(devStates,seed);

	kmodes<<<blocks,TPB>>>(phi_d,devStates,3);

	//hipMemcpy(k_h,k_d,Nk*sizeof(double),hipMemcpyDeviceToHost);
	//hipMemcpy(theta_h,theta_d,Nk*sizeof(double),hipMemcpyDeviceToHost);
	//hipMemcpy(phi_h,phi_d,Nk*sizeof(double),hipMemcpyDeviceToHost);

	/*for(int i=0;i<Nk;i++){
		printf("%d\t %f,%f,%f\n",i,k_h[i],theta_h[i],phi_h[i]);
	}*/

	/*hipFree(init_d);
	hipFree(pos_d);
	hipFree(eta_d);*/
	hipFree(k_d);
	hipFree(theta_d);
	hipFree(phi_d);
}

__global__ void setup_kmodes(hiprandState *state,unsigned long seed){
        int idx=threadIdx.x+blockIdx.x*blockDim.x;
        hiprand_init(seed,idx,0,&state[idx]);
}

__global__ void kmodes(double *vec,hiprandState *globalState,int opt){
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	hiprandState localState=globalState[idx];
	if(idx<Nk_d){
		if(opt==1){
			vec[idx]=pow((pow(kmax,3.0)-pow(kmin,3.0))*hiprand_uniform(&localState)+pow(kmin,3.0),1.0/3.0);
			printf("k[%d]=%f\n",idx,vec[idx]);
		}else if(opt==2){
			vec[idx]=acos(1.0-2.0*hiprand_uniform(&localState));
			printf("theta[%d]=%f\n",idx,vec[idx]);
		}else if(opt==3){
			vec[idx]=2.0*pi*hiprand_uniform(&localState);
			printf("phi[%d]=%f\n",idx,vec[idx]);
		}
	}
}

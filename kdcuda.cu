#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
//#include<cuda_runtime.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include"math.h"
#include<iostream>
#include<fstream>
#include<sstream>

#define TPB 256

/*
When using different methods (Euler, RK2 or RK4) there are different memory settings.
Euler:	31 4-Byte registers, 24 Bytes of shared memory per thread. 1080Ti => 100.0% occupancy, 57344 particles simultaneously.
RK2:	37 4-Byte registers, 48 Bytes of shared memory per thread. 1800Ti =>  75.0% occupancy, 43008 particles simultaneously.
RK4:	43 4-Byte registers, 72 Bytes of shared memory per thread. 1080Ti =>  62.5% occupancy, 35840 particles simultaneously.

********************************************************************************
*************    THIS VERSION IS NOT OPTIMIZED    ******************************
********************************************************************************
*/

#define N 10000 // Number of electrons
#define Nk 1 // Number of k-modes
#define Ne 1 // Number of polarizations per k-mode

__constant__ double pi;
__constant__ double q; // electron charge
__constant__ double m; // electron rest mass
__constant__ double hbar; // Planck's constant
__constant__ double c; // velocity of light in vacuum
__constant__ double eps0;
__constant__ double v0; // electron velocity before laser region
__constant__ double sigma; // electron beam standard deviation
__constant__ double sigma_p; // electron beam transverse momentum standard deviation

__constant__ double wL; // Laser frequency
__constant__ double kL; // kL=wL/c
__constant__ double lamL; // lamL=2pi/kL

__constant__ double wR; // ZPF frequency (resonance)
__constant__ double kR; // kR=wR/c
__constant__ double lamR; // lamR=2pi/kR

__constant__ double E0L; // Laser electric field intensity amplitude
__constant__ double D; // Laser beam waist
__constant__ double zimp; // Screen position (origin set right before laser region)
__constant__ double sigmaL; // laser region standard deviation

__constant__ double damping; // Damping rate (harmonic oscillator approximation)
__constant__ double Delta; // thickness of the spherical shell in k-space using resonance frequency
__constant__ double kmin;
__constant__ double kmax;
__constant__ double V; // Estimated total volume of space

__constant__ double dt; // time step necessary to resolve the electron trajectory

__constant__ double xi[Ne]; // Polarization angles for each k-mode (Ne in total): NOT random, allocated in CONSTANT memory for optimization purposes

void onHost();
void onDevice(double *k,double *theta,double *phi,double *eta,double *angles,double *xi,double *init,double *v_init,double *positions);

__global__ void setup_kmodes(hiprandState *state,unsigned long seed);
__global__ void kmodes(double *x,hiprandState *state,int option,int n);
__global__ void paths_euler(double *k,double *angles,double *pos);
__global__ void paths_rk2(double *k,double *angles,double *pos);
__global__ void paths_rk4(double *k,double *angles,double *pos);

__device__ double f(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vy,double const &vz);
__device__ double g(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vz);
__device__ double gL(double const &t,double const &y,double const &z,double const &vz);
__device__ double h(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vy);
__device__ double hL(double const &t,double const &y,double const &z,double const &vy);

int main(){
	onHost();
	return 0;
}

void onHost(){
	FILE *k_vec,*posit=NULL;

	time_t rawtime;
	struct tm*timeinfo;

	time(&rawtime);
	timeinfo=localtime(&rawtime);

	printf("The current time is %s",asctime(timeinfo));
	
	const char* name_k="k-vectors";
	const char* name_p="screen";
	const char* format=".txt";

	char day[10];

	strftime(day, sizeof(day)-1, "%d_%H_%M", timeinfo);

	char strtmp[6];

	char filename_k[512];
	char filename_p[512];

	std::copy(asctime(timeinfo)+4,asctime(timeinfo)+7,strtmp);

	sprintf(filename_k,"%s%s%s%s",name_k,strtmp,day,format);
	sprintf(filename_p,"%s%s%s%s",name_p,strtmp,day,format);

	double *k_h,*theta_h,*phi_h; // Spherical coordinates for each k-mode (Nk in total)
	double *eta_h; // Random phases for the ZPF k-modes (Nk in total) <--- Following Boyer's work
	double *angles_h; // Single vector for the theta, phi and eta random numbers (3Nk in length for optimization purposes)
	double *xi_h; // Polarization angles in host space
	double *init_h; // Initial positions (h indicates host allocation)
	double *v_init_h; // Initial transverse velocities
	double *screen_h; // Single vector for the initial position, initial transverse velocities and final positions (3N in length for optimization purposes)

	k_h=(double*)malloc(Nk*sizeof(double));
	theta_h=(double*)malloc(Nk*sizeof(double));
	phi_h=(double*)malloc(Nk*sizeof(double));

	eta_h=(double*)malloc(Nk*sizeof(double));

	angles_h=(double*)malloc(3*Nk*sizeof(double));

	xi_h=(double*)malloc(Ne*sizeof(double));

	init_h=(double*)malloc(N*sizeof(double));

	v_init_h=(double*)malloc(N*sizeof(double));

	screen_h=(double*)malloc(3*N*sizeof(double));

	onDevice(k_h,theta_h,phi_h,eta_h,angles_h,xi_h,init_h,v_init_h,screen_h);

	k_vec=fopen(filename_k,"w");
	for(int i=0;i<Nk;i++){
		fprintf(k_vec,"%2.8e,%f,%f,%f\n",k_h[i],angles_h[i],angles_h[Nk+i],angles_h[2*Nk+i]);
	}
	fclose(k_vec);

	posit=fopen(filename_p,"w");
	for(int i=0;i<N;i++){
		fprintf(posit,"%2.6e,%2.6e,%2.6e\n",screen_h[i],screen_h[N+i],screen_h[2*N+i]);
	}
	fclose(posit);

	free(k_h);
	free(theta_h);
	free(phi_h);
	free(eta_h);
	free(angles_h);
	free(xi_h);
	free(init_h);
	free(v_init_h);
	free(screen_h);
}

void onDevice(double *k_h,double *theta_h,double *phi_h,double *eta_h,double *angles_h,double *xi_h,double *init_h,double *v_init_h,double *screen_h){
	unsigned int blocks=(Nk+TPB-1)/TPB;

	double pi_h=3.1415926535;
	double q_h=1.6e-19;
	double m_h=9.10938356e-31;
	double hbar_h=1.0545718e-34;
//	double hbar_h=0; // uncomment this line to see classical results
	double c_h=299792458.0;
	double eps0_h=8.85e-12;
	double v0_h=1.1e7;
	double fwhm_h=25e-6;
	double sigma_h=fwhm_h/(2.0*sqrt(2.0*log(2.0)));

	double lamL_h=532e-9;
	double kL_h=2*pi_h/lamL_h;
	double wL_h=kL_h*c_h;

	double sigma_p_h=4.0*pi_h*(1.0545718e-34)/(lamL_h*sqrt(2.0*log(2.0)));

	double lamR_h=lamL_h;
//	double lamR_h=2*pi_h*hbar_h/(m_h*v0_h);
	double kR_h=2*pi_h/lamR_h;
	double wR_h=kR_h*c_h;

	double E0L_h=1.7777777e8;
	double D_h=125e-6;
	double zimp_h=24e-2+D_h;
	double sigmaL_h=26e-6;

	double damping_h=6.245835e-24;
	double Delta_h=9e7*damping_h*pow(wR_h,2.0);
	double kmin_h=(wR_h-Delta_h/2.0)/c_h;
	double kmax_h=(wR_h+Delta_h/2.0)/c_h;
	double Vk_h=4.0*pi_h*(pow(kmax_h,3.0)-pow(kmin_h,3.0))/3.0;
	double V_h=pow(2.0*pi_h,3.0)*Nk/Vk_h;

	double dt_h=pi_h/(1.5*wR_h);
//	double dt_h=1.0/(0.1*wR_h);

	hipMemcpyToSymbol(HIP_SYMBOL(pi),&pi_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(q),&q_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(m),&m_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(hbar),&hbar_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c),&c_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(eps0),&eps0_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(v0),&v0_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sigma),&sigma_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(lamL),&lamL_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kL),&kL_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(wL),&wL_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(sigma_p),&sigma_p_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(lamR),&lamR_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kR),&kR_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(wR),&wR_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(E0L),&E0L_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(D),&D_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(zimp),&zimp_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sigmaL),&sigmaL_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(damping),&damping_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(Delta),&Delta_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kmin),&kmin_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(kmax),&kmax_h,sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(V),&V_h,sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(dt),&dt_h,sizeof(double));

	/* Polarization modes allocation (in CONSTANT memory) */
	for(int i=0;i<Ne;i++){
		xi_h[i]=i*2*pi_h/Ne;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(xi),xi_h,Ne*sizeof(double));

	float elapsedTime; // Variables to record execution times
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double *k_d,*theta_d,*phi_d;
	double *eta_d;
	double *angles_d;
	double *init_d; // Vectors in Device (d indicates device allocation)
	double *v_init_d;
	double *screen_d;

	printf("Number of particles (N): %d\n",N);
	if(hbar_h>0.0){
		printf("Quantum version of the KD effect\n");
		printf("wR=%2.6e rad/s\n",wR_h);
		printf("Delta=%2.6e\n",Delta_h);
		printf("kmin=%2.6e\n",kmin_h);
		printf("kmax=%2.6e\n",kmax_h);
		printf("Vk=%2.6e\n",Vk_h);
		printf("V=%2.6e\n",V_h);
	}else printf("Classical version of the KD effect\n");
	printf("E0L=%2.6e V/m\n",E0L_h);
	printf("dt=%2.6e s\n",dt_h);
	printf("Number of k-modes (Nk): %d\n",Nk);
	printf("Number of polarizations (Ne): %d\n",Ne);
	printf("Threads per block: %d\n",TPB);
	printf("Number of blocks (k-modes): %d\n",blocks);

	hipMalloc((void**)&k_d,Nk*sizeof(double));
	hipMalloc((void**)&theta_d,Nk*sizeof(double));
	hipMalloc((void**)&phi_d,Nk*sizeof(double));

	hipMalloc((void**)&eta_d,Nk*sizeof(double));

	hipMalloc((void**)&angles_d,3*Nk*sizeof(double));

	hipMalloc((void**)&init_d,N*sizeof(double));

	hipMalloc((void**)&v_init_d,N*sizeof(double));

	hipMalloc((void**)&screen_d,3*N*sizeof(double));

	/* Randomly generated k-modes inside the spherical shell */

	hipEventRecord(start,0);

	hiprandState *devStates_kmodes;
        hipMalloc(&devStates_kmodes,Nk*sizeof(hiprandState));

	//k
	srand(time(0));
	int seed=rand(); //Setting up the seeds
	setup_kmodes<<<blocks,TPB>>>(devStates_kmodes,seed);

	kmodes<<<blocks,TPB>>>(k_d,devStates_kmodes,1,Nk);

	//theta
	kmodes<<<blocks,TPB>>>(theta_d,devStates_kmodes,2,Nk);

	//phi
	kmodes<<<blocks,TPB>>>(phi_d,devStates_kmodes,3,Nk);

	hipMemcpy(k_h,k_d,Nk*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(theta_h,theta_d,Nk*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(phi_h,phi_d,Nk*sizeof(double),hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Random k-modes succesfully generated in %6.4f ms\n",elapsedTime);

	/* Randomly generated phases for the CPC modes */

	hipEventRecord(start,0);

	hiprandState *devStates_eta;
	hipMalloc(&devStates_eta,Nk*sizeof(hiprandState));

	blocks=(Nk+TPB-1)/TPB;
	printf("Number of blocks (phases): %d\n",blocks);

	//eta
	srand(time(NULL));
	seed=rand(); //Settin up seeds
	setup_kmodes<<<blocks,TPB>>>(devStates_eta,seed);

	kmodes<<<blocks,TPB>>>(eta_d,devStates_eta,6,Nk);

	hipMemcpy(eta_h,eta_d,Nk*sizeof(double),hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Random ZPF phases succesfully generated in %6.4f ms\n",elapsedTime);

	/* Making a single vector for theta, phi and eta (reduces the size of memory, one double pointer instead of three) */
	
	for(int i=0;i<Nk;i++){
		angles_h[i]=theta_h[i];
		angles_h[Nk+i]=phi_h[i];
		angles_h[2*Nk+i]=eta_h[i];
		//angles_h[3*Nk+i]=eta_h[i+Nk];
	}

	hipFree(theta_d);
	hipFree(phi_d);
	hipFree(eta_d);

	hipMemcpy(angles_d,angles_h,3*Nk*sizeof(double),hipMemcpyHostToDevice);

	/* Initial positions and transverse momentum*/

	hipEventRecord(start,0);

	hiprandState *devStates_init;
	hipMalloc(&devStates_init,N*sizeof(hiprandState));

	blocks=(N+TPB-1)/TPB;
	printf("Number of blocks (paths): %d\n",blocks);

	srand(time(NULL));
	seed=rand();
	setup_kmodes<<<blocks,TPB>>>(devStates_init,seed);

	kmodes<<<blocks,TPB>>>(init_d,devStates_init,4,N);
	hipMemcpy(init_h,init_d,N*sizeof(double),hipMemcpyDeviceToHost);

	kmodes<<<blocks,TPB>>>(v_init_d,devStates_init,5,N);
	hipMemcpy(v_init_h,v_init_d,N*sizeof(double),hipMemcpyDeviceToHost);

	/* Making a single vector for the initial and final positions (reduces the size of memory, one double pointer instead of two) */

	for(int i=0;i<N;i++){
		screen_h[i]=init_h[i];
		screen_h[N+i]=v_init_h[i];
		screen_h[2*N+i]=0.0;
	}

	hipMemcpy(screen_d,screen_h,3*N*sizeof(double),hipMemcpyHostToDevice);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Positions vector initialized in %6.4f ms\n",elapsedTime);

	hipEventRecord(start,0);

	//paths_euler<<<blocks,TPB>>>(k_d,angles_d,screen_d);
	//paths_rk2<<<blocks,TPB>>>(k_d,angles_d,screen_d);
	paths_rk4<<<blocks,TPB>>>(k_d,angles_d,screen_d);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	//printf("Paths computed using Euler method in %6.4f hours\n",elapsedTime*1e-3/3600.0);
	//printf("Paths computed using RK2 method in %6.4f hours\n",elapsedTime*1e-3/3600.0);
	printf("Paths computed using RK4 method in %6.4f hours\n",elapsedTime*1e-3/3600.0);
	printf("------------------------------------------------------------\n");

	hipMemcpy(screen_h,screen_d,3*N*sizeof(double),hipMemcpyDeviceToHost);

	hipFree(devStates_kmodes);
	hipFree(devStates_eta);
	hipFree(devStates_init);
	hipFree(k_d);
	hipFree(angles_d);
	hipFree(init_d);
	hipFree(v_init_d);
	hipFree(screen_d);
}

__global__ void setup_kmodes(hiprandState *state,unsigned long seed){
        int idx=threadIdx.x+blockIdx.x*blockDim.x;
        hiprand_init(seed,idx,0,&state[idx]);
}

__global__ void kmodes(double *vec,hiprandState *globalState,int opt,int n){
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	hiprandState localState=globalState[idx];
	if(idx<n){
		if(opt==1){ // Random radii
			if(idx==0){
				vec[idx]=kL/2.0;
			}else if(idx==1){
				vec[idx]=kL;
			}else if(idx==2){
				vec[idx]=3.0*kL/2.0;
			}else{
				vec[idx]=pow((pow(kmax,3.0)-pow(kmin,3.0))*hiprand_uniform(&localState)+pow(kmin,3.0),1.0/3.0);
			}
			//vec[idx]=pow((pow(kmax,3.0)-pow(kmin,3.0))*curand_uniform(&localState)+pow(kmin,3.0),1.0/3.0);
		}else if(opt==2){ // Random polar angles
			if(idx==0||idx==1||idx==2){
				vec[idx]=pi/2.0;
			}else{
				vec[idx]=acos(1.0-2.0*hiprand_uniform(&localState));
			}
			//vec[idx]=acos(1.0-2.0*curand_uniform(&localState));
		}else if(opt==3){ // Random azimuthal angles
			if(idx==0||idx==1||idx==2){
				vec[idx]=pi/2.0;
			}else{
				vec[idx]=2.0*pi*hiprand_uniform(&localState);
			}
			//vec[idx]=2.0*pi*curand_uniform(&localState);
		}else if(opt==4){
			vec[idx]=sigma*hiprand_normal(&localState); // Random initial positions
		}else if(opt==5){
			vec[idx]=sigma_p*hiprand_normal(&localState); // Random initial transverse momentum
		}else if(opt==6){
			if(idx==0||idx==1||idx==2){
				vec[idx]=0.001;
			}else{
				vec[idx]=2.0*pi*hiprand_uniform(&localState); // Random constant phases
			}
			//vec[idx]=2.0*pi*curand_uniform(&localState);
		}
		globalState[idx]=localState; // Update current seed state
	}
}

__global__ void paths_euler(double *k,double *angles,double *pos){
	unsigned int idx=threadIdx.x+blockIdx.x*TPB;
	
	__shared__ double vxnn[TPB];
	__shared__ double vynn[TPB];
	__shared__ double vznn[TPB];

	if(idx<N){
		double tn=0.0;
		double xn=0.0;
		double yn=pos[idx];
		double zn=0.0;

		double vxn=0.0;
		double vyn=pos[N+idx];
		__syncthreads();
		double vzn=v0;

		vxnn[threadIdx.x]=0.0;
		vynn[threadIdx.x]=0.0;
		vznn[threadIdx.x]=0.0;

		while(zn<=D){
			for(int i=0;i<Nk;i++){
				for(int j=0;j<Ne;j++){
					__syncthreads();
					vxnn[threadIdx.x]=vxnn[threadIdx.x]+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vyn,vzn); // vxnn represents here the total ZPF force in x (recycled variable)
					__syncthreads();
					vynn[threadIdx.x]=vynn[threadIdx.x]+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vxn,vzn); // k1vy represents here the total ZPF force in y
					__syncthreads();
					vznn[threadIdx.x]=vznn[threadIdx.x]+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vxn,vyn); // k1vz represents here the total ZPF force in z
				}
			}
			vynn[threadIdx.x]=vynn[threadIdx.x]+gL(tn,yn,zn,vzn);
			vznn[threadIdx.x]=vznn[threadIdx.x]+hL(tn,yn,zn,vyn);

			__syncthreads();
			vxnn[threadIdx.x]=vxn+dt*vxnn[threadIdx.x];
			__syncthreads();
			vynn[threadIdx.x]=vyn+dt*vynn[threadIdx.x];
			__syncthreads();
			vznn[threadIdx.x]=vzn+dt*vznn[threadIdx.x];
			__syncthreads();
			tn=tn+dt;
			__syncthreads();
			xn=xn+dt*vxn;
			__syncthreads();
			yn=yn+dt*vyn;
			__syncthreads();
			zn=zn+dt*vzn;

			vxn=vxnn[threadIdx.x];
			vyn=vynn[threadIdx.x];
			vzn=vznn[threadIdx.x];
		}
		__syncthreads();
		pos[2*N+idx]=yn+(zimp-D)*vyn/vzn;
	}
}
__global__ void paths_rk2(double *k,double *angles,double *pos){
	unsigned int idx=threadIdx.x+blockIdx.x*TPB;

	__shared__ double k1vx[TPB];
	__shared__ double k1vy[TPB];
	__shared__ double k1vz[TPB];
	__shared__ double k2vx[TPB];
	__shared__ double k2vy[TPB];
	__shared__ double k2vz[TPB];

	if(idx<N){
		double tn=0.0;
		double xn=0.0;
		double yn=pos[idx];
		double zn=0.0;

		double vxn=0.0;
		double vyn=pos[N+idx];
		__syncthreads();
		double vzn=v0;

		double vxnn=0.0;
		double vynn=0.0;
		double vznn=0.0;

		while(zn<=D){ // Only laser region. After the particle leaves it, the final position is extrapolated
			k1vx[threadIdx.x]=0.0;
			k1vy[threadIdx.x]=0.0;
			k1vz[threadIdx.x]=0.0;
			k2vx[threadIdx.x]=0.0;
			k2vy[threadIdx.x]=0.0;
			k2vz[threadIdx.x]=0.0;

			for(int i=0;i<Nk;i++){
				for(int j=0;j<Ne;j++){
					__syncthreads();
					k1vx[threadIdx.x]=k1vx[threadIdx.x]+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vyn,vzn); // k1vx represents here the total ZPF force in x
					__syncthreads();
					k1vy[threadIdx.x]=k1vy[threadIdx.x]+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vxn,vzn); // k1vy represents here the total ZPF force in y
					__syncthreads();
					k1vz[threadIdx.x]=k1vz[threadIdx.x]+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vxn,vyn); // k1vz represents here the total ZPF force in z
				}
			}

			k1vy[threadIdx.x]=k1vy[threadIdx.x]+gL(tn,yn,zn,vzn); // Laser contribution to the total force in y
			k1vz[threadIdx.x]=k1vz[threadIdx.x]+hL(tn,yn,zn,vyn); // Laser contribution to the total force in z

			for(int i=0;i<Nk;i++){
				for(int j=0;j<Ne;j++){
					__syncthreads();
					k2vx[threadIdx.x]=k2vx[threadIdx.x]+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn+dt,xn+dt*vxn,yn+dt*vyn,zn+dt*vzn,vyn+dt*k1vy[threadIdx.x],vzn+dt*k1vz[threadIdx.x]); // k2vx represents here the total ZPF force in x
					__syncthreads();
					k2vy[threadIdx.x]=k2vy[threadIdx.x]+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn+dt,xn+dt*vxn,yn+dt*vyn,zn+dt*vzn,vxn+dt*k1vx[threadIdx.x],vzn+dt*k1vz[threadIdx.x]); // k2vy represents here the total ZPF force in y
					__syncthreads();
					k2vz[threadIdx.x]=k2vz[threadIdx.x]+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn+dt,xn+dt*vxn,yn+dt*vyn,zn+dt*vzn,vxn+dt*k1vx[threadIdx.x],vyn+dt*k1vy[threadIdx.x]); // k2vz represents here the total ZPF force in z
				}
			}

			__syncthreads();
			k2vy[threadIdx.x]=k2vy[threadIdx.x]+gL(tn+dt,yn+dt*vyn,zn+dt*vzn,vzn+dt*k1vz[threadIdx.x]); // Laser contribution to the total force in y
			__syncthreads();
			k2vz[threadIdx.x]=k2vz[threadIdx.x]+hL(tn+dt,yn+dt*vyn,zn+dt*vzn,vyn+dt*k1vy[threadIdx.x]); // Laser contribution to the total force in z

			__syncthreads();
			vxnn=vxn+dt*(k1vx[threadIdx.x]+k2vx[threadIdx.x])/2.0;
			__syncthreads();
			vynn=vyn+dt*(k1vy[threadIdx.x]+k2vy[threadIdx.x])/2.0;
			__syncthreads();
			vznn=vzn+dt*(k1vz[threadIdx.x]+k2vz[threadIdx.x])/2.0;

			__syncthreads();
			xn=xn+dt*(vxn+vxnn)/2.0;
			__syncthreads();
			yn=yn+dt*(vyn+vynn)/2.0;
			__syncthreads();
			zn=zn+dt*(vzn+vznn)/2.0;
		}
		__syncthreads();
		pos[2*N+idx]=yn+(zimp-D)*vyn/vzn;
	}
}
__global__ void paths_rk4(double *k,double *angles,double *pos){
	unsigned int idx=threadIdx.x+blockIdx.x*TPB;

	__shared__ double k1vx[TPB];
	__shared__ double k1vy[TPB];
	__shared__ double k1vz[TPB];
	
	__shared__ double k2vx[TPB];
	__shared__ double k2vy[TPB];
	__shared__ double k2vz[TPB];
	
	__shared__ double k3vx[TPB];
	__shared__ double k3vy[TPB];
	__shared__ double k3vz[TPB];

	if(idx<N){
		double tn=0.0;

		double xn=0.0;
		double yn=pos[idx];
		double zn=0.0;

		double vxn=0.0;
		double vyn=pos[N+idx];;
		__syncthreads();
		double vzn=v0;

		double k1x;
		double k1y;
		double k1z;

		double k2x;
		double k2y;
		double k2z;

		double k3x;
		double k3y;
		double k3z;

		double k4x;
		double k4y;
		double k4z;

		double k4vx;
		double k4vy;
		double k4vz;

		while(zn<=D){
			k1x=0.0;
			k1y=0.0;
			k1z=0.0;

			k2x=0.0;
			k2y=0.0;
			k2z=0.0;
			
			k3x=0.0;
			k3y=0.0;
			k3z=0.0;

			k4x=0.0;
			k4y=0.0;
			k4z=0.0;

			k1vx[threadIdx.x]=0.0;
			k1vy[threadIdx.x]=0.0;
			k1vz[threadIdx.x]=0.0;

			k2vx[threadIdx.x]=0.0;
			k2vy[threadIdx.x]=0.0;
			k2vz[threadIdx.x]=0.0;

			k3vx[threadIdx.x]=0.0;
			k3vy[threadIdx.x]=0.0;
			k3vz[threadIdx.x]=0.0;

			k4vx=0.0;
			k4vy=0.0;
			k4vz=0.0;

			for(int i=0;i<Nk;i++){ // k1
				for(int j=0;i<Ne;i++){
					__syncthreads();
					k1vx[threadIdx.x]=k1vx[threadIdx.x]+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vyn,vzn); // k1vx represents here the total ZPF force in x
					__syncthreads();
					k1vy[threadIdx.x]=k1vy[threadIdx.x]+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vxn,vzn); // k1vy represents here the total ZPF force in y
					__syncthreads();
					k1vz[threadIdx.x]=k1vz[threadIdx.x]+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn,yn,zn,vxn,vyn); // k1vz represents here the total ZPF force in z
				}
			}
			__syncthreads();
			k1vy[threadIdx.x]=k1vy[threadIdx.x]+gL(tn,yn,zn,vzn); // Laser contribution to the total force in y
			__syncthreads();
			k1vz[threadIdx.x]=k1vz[threadIdx.x]+hL(tn,yn,zn,vyn); // Laser contribution to the total force in z

			__syncthreads();
			tn=tn+dt/2.0;

			k1x=vxn;
			k1y=vyn;
			k1z=vzn;
			
			for(int i=0;i<Nk;i++){ // k2
				for(int j=0;j<Ne;j++){
					__syncthreads();
					k2vx[threadIdx.x]=k2vx[threadIdx.x]+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k1x/2.0,yn+dt*k1y/2.0,zn+dt*k1z/2.0,vyn+dt*k1vy[threadIdx.x]/2.0,vzn+dt*k1vz[threadIdx.x]/2.0); // k2vx represents here the total force in x
					__syncthreads();
					k2vy[threadIdx.x]=k2vy[threadIdx.x]+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k1x/2.0,yn+dt*k1y/2.0,zn+dt*k1z/2.0,vxn+dt*k1vx[threadIdx.x]/2.0,vzn+dt*k1vz[threadIdx.x]/2.0); // k2vy represents here the total force in y
					__syncthreads();
					k2vz[threadIdx.x]=k2vz[threadIdx.x]+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k1x/2.0,yn+dt*k1y/2.0,zn+dt*k1z/2.0,vxn+dt*k1vx[threadIdx.x]/2.0,vyn+dt*k1vy[threadIdx.x]/2.0); // k2vz represents here the total force in z
				}
			}
			__syncthreads();
			k2vy[threadIdx.x]=k2vy[threadIdx.x]+gL(tn,yn+dt*k1y/2.0,zn+dt*k1z/2.0,vzn+dt*k1vz[threadIdx.x]/2.0); // Laser contribution to the total force in y
			__syncthreads();
			k2vz[threadIdx.x]=k2vz[threadIdx.x]+hL(tn,yn+dt*k1y/2.0,zn+dt*k1z/2.0,vyn+dt*k1vy[threadIdx.x]/2.0); // Laser contribution to the total force in z

			__syncthreads();
			k2x=vxn+dt*k1vx[threadIdx.x]/2.0;
			__syncthreads();
			k2y=vyn+dt*k1vy[threadIdx.x]/2.0;
			__syncthreads();
			k2z=vzn+dt*k1vz[threadIdx.x]/2.0;

			for(int i=0;i<Nk;i++){ // k3
				for(int j=0;j<Ne;j++){
					__syncthreads();
					k3vx[threadIdx.x]=k3vx[threadIdx.x]+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k2x/2.0,yn+dt*k2y/2.0,zn+dt*k2z/2.0,vyn+dt*k2vy[threadIdx.x]/2.0,vzn+dt*k2vz[threadIdx.x]/2.0); // k3vx represents here the total force in x
					__syncthreads();
					k3vy[threadIdx.x]=k3vy[threadIdx.x]+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k2x/2.0,yn+dt*k2y/2.0,zn+dt*k2z/2.0,vxn+dt*k2vx[threadIdx.x]/2.0,vzn+dt*k2vz[threadIdx.x]/2.0); // k3vy represents here the total force in y
					__syncthreads();
					k3vz[threadIdx.x]=k3vz[threadIdx.x]+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k2x/2.0,yn+dt*k2y/2.0,zn+dt*k2z/2.0,vxn+dt*k2vx[threadIdx.x]/2.0,vyn+dt*k2vy[threadIdx.x]/2.0); // k3vz represents here the total force in z
				}
			}
			__syncthreads();
			k3vy[threadIdx.x]=k3vy[threadIdx.x]+gL(tn,yn+dt*k2y/2.0,zn+dt*k2z/2.0,vzn+dt*k2vz[threadIdx.x]/2.0); // Laser contribution to the total force in y
			__syncthreads();
			k3vz[threadIdx.x]=k3vz[threadIdx.x]+hL(tn,yn+dt*k2y/2.0,zn+dt*k2z/2.0,vyn+dt*k2vy[threadIdx.x]/2.0); // Laser contribution to the total force in z

			__syncthreads();
			tn=tn+dt/2.0;

			__syncthreads();
			k3x=vxn+dt*k2vx[threadIdx.x]/2.0;
			__syncthreads();
			k3y=vyn+dt*k2vy[threadIdx.x]/2.0;
			__syncthreads();
			k3z=vzn+dt*k2vz[threadIdx.x]/2.0;

			for(int i=0;i<Nk;i++){ // k4
				for(int j=0;j<Ne;j++){
					__syncthreads();
					k4vx=k4vx+f(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k3x,yn+dt*k3y,zn+dt*k3z,vyn+dt*k3vy[threadIdx.x],vzn+dt*k3vz[threadIdx.x]); // k4vx represents here the total force in x
					__syncthreads();
					k4vy=k4vy+g(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k3x,yn+dt*k3y,zn+dt*k3z,vxn+dt*k3vx[threadIdx.x],vzn+dt*k3vz[threadIdx.x]); // k4vy represents here the total force in y
					__syncthreads();
					k4vz=k4vz+h(k[i],angles[i],angles[Nk+i],angles[2*Nk+i],xi[j],tn,xn+dt*k3x,yn+dt*k3y,zn+dt*k3z,vxn+dt*k3vx[threadIdx.x],vyn+dt*k3vy[threadIdx.x]); // k4vz represents here the total force in z
				}
			}
			k4vy=k4vy+gL(tn,yn+dt*k3y,zn+dt*k3z,vzn+dt*k3vz[threadIdx.x]); // Laser contribution to the total force in y
			k4vz=k4vz+hL(tn,yn+dt*k3y,zn+dt*k3z,vyn+dt*k3vy[threadIdx.x]); // Laser contribution to the total force in z

			__syncthreads();
			xn=xn+dt*(k1x+2.0*k2x+2.0*k3x+k4x)/6.0;
			__syncthreads();
			yn=yn+dt*(k1y+2.0*k2y+2.0*k3y+k4y)/6.0;
			__syncthreads();
			zn=zn+dt*(k1z+2.0*k2z+2.0*k3z+k4z)/6.0;

			__syncthreads();
			vxn=vxn+dt*(k1vx[threadIdx.x]+2.0*k2vx[threadIdx.x]+2.0*k3vx[threadIdx.x]+k4vx)/6.0;
			__syncthreads();
			vyn=vyn+dt*(k1vy[threadIdx.x]+2.0*k2vy[threadIdx.x]+2.0*k3vy[threadIdx.x]+k4vy)/6.0;
			__syncthreads();
			vzn=vzn+dt*(k1vz[threadIdx.x]+2.0*k2vz[threadIdx.x]+2.0*k3vz[threadIdx.x]+k4vz)/6.0;
		}
		__syncthreads();
		pos[2*N+idx]=yn+(zimp-D)*vyn/vzn;
	}
}
/*
__device__ double f(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vy,double const &vz){ // ZPF, x-component (Wayne-Herman version)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	__syncthreads();
	double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;

	__syncthreads();
	double E0=10.0*sqrt(hbar*w/(eps0*V));

	__syncthreads();
	return q*E0*(cos(phi1)+cos(phi2))*(cos(theta)*cos(phi)*cos(xi)-sin(phi)*sin(xi))/m+q*E0*(cos(phi1)-cos(phi2))*(sin(theta)*sin(xi)*vy+(cos(theta)*sin(phi)*sin(xi)-cos(phi)*cos(xi))*vz)/(m*c);
}
*/
/*
__device__ double f(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vy,double const &vz){ // ZPF, x-component (WH+multiple Fourier modes)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	__syncthreads();
	double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;

	double E0;
	if(k==kR/2.0){
		__syncthreads();
		E0=sqrt(2.0/(3.0*pi))*E0L;
	}else if(k==kR){
		__syncthreads();
		E0=sqrt(1.0/2.0)*E0L;
	}else if(k==3.0*kR/2.0){
		__syncthreads();
		E0=sqrt(6.0/(5.0*pi))*E0L;
	}else{
		__syncthreads();
		E0=sqrt(hbar*w/(eps0*V));
	}
	return 0.0;
}
*/

__device__ double f(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vy,double const &vz){ // ZPF, x-component (Single explicit Fourier mode)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*y+eta;
	__syncthreads();
	double phi2=w*t+k*y+eta;

	double E0;
	if(k==kR/2.0){
		__syncthreads();
		E0=sqrt(2.0/(3.0*pi))*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0))); // SHOULD I USE LASER GAUSSIAN PROFILE HERE OR NOT?
	}else if(k==kR){
		__syncthreads();
		E0=sqrt(1.0/2.0)*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else if(k==3.0*kR/2.0){
		__syncthreads();
		E0=sqrt(6.0/(5.0*pi))*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else{
		__syncthreads();
		E0=sqrt(hbar*w/(eps0*V));
	}
	//__syncthreads();
	//return q*E0*(cos(phi1)-cos(phi2))*(sin(theta)*sin(xi)*vy+(cos(theta)*sin(phi)*sin(xi)-cos(phi)*cos(xi))*vz)/(m*c);
	return 0.0;
}

/*
__device__ double g(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vz){ // ZPF, y-component (Wayne-Herman version)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	__syncthreads();
	double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;

	__syncthreads();
	double E0=10.0*sqrt(hbar*w/(eps0*V));

	__syncthreads();
	return q*E0*(cos(phi1)+cos(phi2))*(cos(theta)*sin(phi)*cos(xi)+cos(phi)*sin(xi))/m-q*E0*(cos(phi1)-cos(phi2))*(sin(theta)*sin(xi)*vx+(cos(theta)*cos(phi)*sin(xi)+sin(phi)*cos(xi))*vz)/(m*c);
}
*/
/*
__device__ double g(double &kv,double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vz){ // ZPF, y-component (WH+multiple Fourier modes)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	__syncthreads();
	double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;

	double E0;
	if(k==kR/2.0){
		__syncthreads();
		E0=sqrt(2.0/(3.0*pi))*E0L;
	}else if(k==kR){
		__syncthreads();
		E0=(1.0/sqrt(2.0))*E0L;
	}else if(k==3.0*kR/2.0){
		__syncthreads();
		E0=sqrt(6.0/(5.0*pi))*E0L;
	}else{
		__syncthreads();
		E0=sqrt(hbar*w/(eps0*V));
	}
	__syncthreads();
	return -q*E0*(cos(phi1)-cos(phi2))*vz/(m*c);
}
*/

__device__ double g(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vz){ // ZPF, y-component (Single explicit Fourier mode)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*y+eta;
	__syncthreads();
	double phi2=w*t+k*y+eta;

	double E0;
	if(k==kR/2.0){
		__syncthreads();
		E0=sqrt(2.0/(3.0*pi))*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else if(k==kR){
		__syncthreads();
		E0=sqrt(1.0/2.0)*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else if(k==3.0*kR/2.0){
		__syncthreads();
		E0=sqrt(6.0/(5.0*pi))*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else{
		__syncthreads();
		E0=sqrt(hbar*w/(eps0*V));
	}
	__syncthreads();
	return -q*E0*(cos(phi1)-cos(phi2))*vz/(m*c);
}
__device__ double gL(double const &t,double const &y,double const &z,double const &vz){ // Laser region, y-component
	__syncthreads();
	double phi1=wL*t-kL*y;
	__syncthreads();
	double phi2=wL*t+kL*y;

	__syncthreads();
	double E0=E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	
	__syncthreads();
	return q*E0*(cos(phi1)-cos(phi2))*vz/(m*c);
}
/*
__device__ double h(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vy){ // ZPF, z-component (Wayne-Herman version)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	__syncthreads();
	double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	
	__syncthreads();
	double E0=10.0*sqrt(hbar*w/(eps0*V));

	__syncthreads();
	return -q*E0*(cos(phi1)+cos(phi2))*(sin(theta)*cos(xi))/m+q*E0*(cos(phi1)-cos(phi2))*((cos(phi)*cos(xi)-cos(theta)*sin(phi)*sin(xi))*vx+(sin(phi)*cos(xi)+cos(theta)*cos(phi)*sin(xi))*vy)/(m*c);
}
*/
/*
__device__ double h(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vy){ // ZPF, z-component (WH+multiple Fourier modes)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	__syncthreads();
	double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	
	double E0
	if(k==kR/2.0){
		__syncthreads();
		E0=sqrt(2.0/(3.0*pi))*E0L;
	}else if(k==kR){
		__syncthreads();
		E0=(1/sqrt(2.0))*E0L;
	}else if(k==3.0*kR/2.0){
		__syncthreads();
		E0=sqrt(6.0/(5.0*pi))*E0L;
	}else{
		__syncthreads();
		E0=sqrt(hbar*w/(eps0*V));
	}
	__syncthreads();
	return -q*E0*(cos(phi1)+cos(phi2))/m+q*E0*(cos(phi1)-cos(phi2))*vy/(m*c);
}
*/

__device__ double h(double const &k,double const &theta,double const &phi,double const &eta,double &xi,double const &t,double const &x,double const &y,double const &z,double const &vx,double const &vy){ // ZPF, z-component (Single explicit Fourier mode)
	__syncthreads();
	double w=k*c;

	__syncthreads();
	//double phi1=w*t-k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	double phi1=w*t-k*y+eta;
	__syncthreads();
	//double phi2=w*t+k*(sin(theta)*cos(phi)*x+sin(theta)*sin(phi)*y+cos(theta)*z)+eta;
	double phi2=w*t+k*y+eta;
	
	double E0;
	if(k==kR/2.0){
		__syncthreads();
		E0=sqrt(2.0/(3.0*pi))*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else if(k==kR){
		__syncthreads();
		E0=sqrt(1.0/2.0)*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else if(k==3.0*kR/2.0){
		__syncthreads();
		E0=sqrt(6.0/(5.0*pi))*E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	}else{
		__syncthreads();
		E0=sqrt(hbar*w/(eps0*V));
	}
	__syncthreads();
	return -q*E0*(cos(phi1)+cos(phi2))/m+q*E0*(cos(phi1)-cos(phi2))*vy/(m*c);
}

__device__ double hL(double const &t,double const &y,double const &z,double const &vy){ // Laser region, z-component
	__syncthreads();
	double phi1=wL*t-kL*y;
	__syncthreads();
	double phi2=wL*t+kL*y;

	__syncthreads();
	double E0=E0L*exp(-pow(z-D/2.0,2.0)/(2.0*pow(sigmaL,2.0)));
	
	__syncthreads();
	return q*E0*(cos(phi1)+cos(phi2))/m-q*E0*(cos(phi1)-cos(phi2))*vy/(m*c);
}
